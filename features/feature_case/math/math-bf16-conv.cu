// ====---------- math-bf16-conv.cu---------- *- CUDA -* ------------------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===---------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <vector>

#include "hip/hip_bf16.h"

using namespace std;

typedef pair<float2, int> f2i_pair;
typedef pair<float, int> fi_pair;
typedef pair<__hip_bfloat162, int> bf162i_pair;
typedef pair<__hip_bfloat16, int> bf16i_pair;

int passed = 0;
int failed = 0;

void check(bool IsPassed) {
  if (IsPassed) {
    cout << " ---- passed" << endl;
    passed++;
  } else {
    cout << " ---- failed" << endl;
    failed++;
  }
}

void checkResult(const string &FuncName, const vector<float> &Inputs,
                 const float &Expect, const float &Result,
                 const int precision) {
  cout << FuncName << "(" << Inputs[0] << "";
  for (size_t i = 1; i < Inputs.size(); ++i) {
    cout << ", " << Inputs[i];
  }
  cout << ") = " << fixed << setprecision(precision) << Result << " (expect "
       << Expect - pow(10, -precision) << " ~ " << Expect + pow(10, -precision)
       << ")";
  cout.unsetf(ios::fixed);
  check(abs(Result - Expect) < pow(10, -precision));
}

void checkResult(const string &FuncName, const vector<float2> &Inputs,
                 const float2 &Expect, const float2 &Result,
                 const int precision) {
  cout << FuncName << "({" << Inputs[0].x << ", " << Inputs[0].y << "}";
  for (size_t i = 1; i < Inputs.size(); ++i) {
    cout << ", {" << Inputs[i].x << ", " << Inputs[i].y << "}";
  }
  cout << ") = " << fixed << setprecision(precision) << "{" << Result.x << ", "
       << Result.y << "} (expect {" << Expect.x - pow(10, -precision) << " ~ "
       << Expect.x + pow(10, -precision) << ", "
       << Expect.y - pow(10, -precision) << " ~ "
       << Expect.y + pow(10, -precision) << ")";
  cout.unsetf(ios::fixed);
  check(abs(Result.x - Expect.x) < pow(10, -precision) &&
        abs(Result.y - Expect.y) < pow(10, -precision));
}

void checkResult(const string &FuncName, const vector<__hip_bfloat16> &Inputs,
                 const __hip_bfloat16 &Expect, const float &Result,
                 const int precision) {
  vector<float> FInputs;
  for (const auto &it : Inputs) {
    FInputs.push_back(__bfloat162float(it));
  }
  float FExpect = __bfloat162float(Expect);
  checkResult(FuncName, FInputs, FExpect, Result, precision);
}

void checkResult(const string &FuncName, const vector<__hip_bfloat162> &Inputs,
                 const float2 &Expect, const float2 &Result,
                 const int precision) {
  vector<float2> FInputs;
  for (const auto &it : Inputs) {
    FInputs.push_back({__bfloat162float(it.x), __bfloat162float(it.y)});
  }
  checkResult(FuncName, FInputs, Expect, Result, precision);
}

void checkResult(const string &FuncName, const vector<float2> &Inputs,
                 const __hip_bfloat162 &Expect, const float2 &Result,
                 const int precision) {
  float2 FExpect{__bfloat162float(Expect.x), __bfloat162float(Expect.y)};
  checkResult(FuncName, Inputs, FExpect, Result, precision);
}

void checkResult(const string &FuncName, const vector<__hip_bfloat162> &Inputs,
                 const __hip_bfloat162 &Expect, const float2 &Result,
                 const int precision) {
  vector<float2> FInputs;
  for (const auto &it : Inputs) {
    FInputs.push_back({__bfloat162float(it.x), __bfloat162float(it.y)});
  }
  checkResult(FuncName, FInputs, Expect, Result, precision);
}

__global__ void setValue(__hip_bfloat16 *Input1, const __hip_bfloat16 Input2) {
  *Input1 = Input2;
}

__global__ void setValue(__hip_bfloat162 *Input1, const __hip_bfloat162 Input2) {
  *Input1 = Input2;
}

__global__ void bFloat1622float2(float *const Result, __hip_bfloat162 Input1) {
  auto ret = __bfloat1622float2(Input1);
  Result[0] = ret.x;
  Result[1] = ret.y;
}

void testBFloat1622float2Cases(
    const vector<pair<__hip_bfloat162, f2i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    bFloat1622float2<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("__bfloat1622float2", {TestCase.first}, TestCase.second.first,
                {Result[0], Result[1]}, TestCase.second.second);
    auto ret = __bfloat1622float2(TestCase.first);
    Result[0] = ret.x;
    Result[1] = ret.y;
    checkResult("(host)__bfloat1622float2", {TestCase.first},
                TestCase.second.first, {Result[0], Result[1]},
                TestCase.second.second);
  }
}

__global__ void bFloat162float(float *const Result, __hip_bfloat16 Input1) {
  *Result = __bfloat162float(Input1);
}

void testBFloat162floatCases(
    const vector<pair<__hip_bfloat16, fi_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    bFloat162float<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("__bfloat162float", {TestCase.first}, TestCase.second.first,
                *Result, TestCase.second.second);
    *Result = __bfloat162float(TestCase.first);
    checkResult("(host)__bfloat162float", {TestCase.first},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void float22bFloat162_rn(float *const Result, float2 Input1) {
  auto ret = __float22bfloat162_rn(Input1);
  Result[0] = __bfloat162float(ret.x);
  Result[1] = __bfloat162float(ret.y);
}

void testFloat22bFloat162_rnCases(
    const vector<pair<float2, bf162i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    float22bFloat162_rn<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("__float22bfloat162_rn", {TestCase.first},
                TestCase.second.first, {Result[0], Result[1]},
                TestCase.second.second);
    auto ret = __float22bfloat162_rn(TestCase.first);
    Result[0] = __bfloat162float(ret.x);
    Result[1] = __bfloat162float(ret.y);
    checkResult("(host)__float22bfloat162_rn", {TestCase.first},
                TestCase.second.first, {Result[0], Result[1]},
                TestCase.second.second);
  }
}

__global__ void float2bFloat16(float *const Result, float Input1) {
  *Result = __bfloat162float(__float2bfloat16(Input1));
}

void testFloat2bFloat16Cases(const vector<pair<float, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    float2bFloat16<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("__float2bfloat16", {TestCase.first}, TestCase.second.first,
                *Result, TestCase.second.second);
    *Result = __float2bfloat16(TestCase.first);
    checkResult("(host)__float2bfloat16", {TestCase.first},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void ldca(float *const Result, __hip_bfloat16 *Input1) {
  *Result = __ldca(Input1);
}

void testLdcaCases(const vector<pair<__hip_bfloat16, int>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    __hip_bfloat16 *Input;
    hipMallocManaged(&Input, sizeof(*Input));
    setValue<<<1, 1>>>(Input, TestCase.first);
    hipDeviceSynchronize();
    ldca<<<1, 1>>>(Result, Input);
    hipDeviceSynchronize();
    checkResult("__ldca", {TestCase.first}, TestCase.first, *Result,
                TestCase.second);
  }
}

__global__ void ldca(float *const Result, __hip_bfloat162 *Input1) {
  auto ret = __ldca(Input1);
  Result[0] = __bfloat162float(ret.x);
  Result[1] = __bfloat162float(ret.y);
}

void testLdcaCases(const vector<pair<__hip_bfloat162, int>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    __hip_bfloat162 *Input;
    hipMallocManaged(&Input, sizeof(*Input));
    setValue<<<1, 1>>>(Input, TestCase.first);
    hipDeviceSynchronize();
    ldca<<<1, 1>>>(Result, Input);
    hipDeviceSynchronize();
    checkResult("__ldca", {TestCase.first}, TestCase.first,
                {Result[0], Result[1]}, TestCase.second);
  }
}

__global__ void ldcg(float *const Result, __hip_bfloat16 *Input1) {
  *Result = __ldcg(Input1);
}

void testLdcgCases(const vector<pair<__hip_bfloat16, int>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    __hip_bfloat16 *Input;
    hipMallocManaged(&Input, sizeof(*Input));
    setValue<<<1, 1>>>(Input, TestCase.first);
    hipDeviceSynchronize();
    ldcg<<<1, 1>>>(Result, Input);
    hipDeviceSynchronize();
    checkResult("__ldcg", {TestCase.first}, TestCase.first, *Result,
                TestCase.second);
  }
}

__global__ void ldcg(float *const Result, __hip_bfloat162 *Input1) {
  auto ret = __ldcg(Input1);
  Result[0] = __bfloat162float(ret.x);
  Result[1] = __bfloat162float(ret.y);
}

void testLdcgCases(const vector<pair<__hip_bfloat162, int>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    __hip_bfloat162 *Input;
    hipMallocManaged(&Input, sizeof(*Input));
    setValue<<<1, 1>>>(Input, TestCase.first);
    hipDeviceSynchronize();
    ldcg<<<1, 1>>>(Result, Input);
    hipDeviceSynchronize();
    checkResult("__ldcg", {TestCase.first}, TestCase.first,
                {Result[0], Result[1]}, TestCase.second);
  }
}

__global__ void ldcs(float *const Result, __hip_bfloat16 *Input1) {
  *Result = __ldcs(Input1);
}

void testLdcsCases(const vector<pair<__hip_bfloat16, int>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    __hip_bfloat16 *Input;
    hipMallocManaged(&Input, sizeof(*Input));
    setValue<<<1, 1>>>(Input, TestCase.first);
    hipDeviceSynchronize();
    ldcs<<<1, 1>>>(Result, Input);
    hipDeviceSynchronize();
    checkResult("__ldcs", {TestCase.first}, TestCase.first, *Result,
                TestCase.second);
  }
}

__global__ void ldcs(float *const Result, __hip_bfloat162 *Input1) {
  auto ret = __ldcs(Input1);
  Result[0] = __bfloat162float(ret.x);
  Result[1] = __bfloat162float(ret.y);
}

void testLdcsCases(const vector<pair<__hip_bfloat162, int>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    __hip_bfloat162 *Input;
    hipMallocManaged(&Input, sizeof(*Input));
    setValue<<<1, 1>>>(Input, TestCase.first);
    hipDeviceSynchronize();
    ldcs<<<1, 1>>>(Result, Input);
    hipDeviceSynchronize();
    checkResult("__ldcs", {TestCase.first}, TestCase.first,
                {Result[0], Result[1]}, TestCase.second);
  }
}

__global__ void ldcv(float *const Result, __hip_bfloat16 *Input1) {
  *Result = __ldcv(Input1);
}

void testLdcvCases(const vector<pair<__hip_bfloat16, int>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    __hip_bfloat16 *Input;
    hipMallocManaged(&Input, sizeof(*Input));
    setValue<<<1, 1>>>(Input, TestCase.first);
    hipDeviceSynchronize();
    ldcv<<<1, 1>>>(Result, Input);
    hipDeviceSynchronize();
    checkResult("__ldcv", {TestCase.first}, TestCase.first, *Result,
                TestCase.second);
  }
}

__global__ void ldcv(float *const Result, __hip_bfloat162 *Input1) {
  auto ret = __ldcv(Input1);
  Result[0] = __bfloat162float(ret.x);
  Result[1] = __bfloat162float(ret.y);
}

void testLdcvCases(const vector<pair<__hip_bfloat162, int>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    __hip_bfloat162 *Input;
    hipMallocManaged(&Input, sizeof(*Input));
    setValue<<<1, 1>>>(Input, TestCase.first);
    hipDeviceSynchronize();
    ldcv<<<1, 1>>>(Result, Input);
    hipDeviceSynchronize();
    checkResult("__ldcv", {TestCase.first}, TestCase.first,
                {Result[0], Result[1]}, TestCase.second);
  }
}

__global__ void ldg(float *const Result, __hip_bfloat16 *Input1) {
  *Result = __ldg(Input1);
}

void testLdgCases(const vector<pair<__hip_bfloat16, int>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    __hip_bfloat16 *Input;
    hipMallocManaged(&Input, sizeof(*Input));
    setValue<<<1, 1>>>(Input, TestCase.first);
    hipDeviceSynchronize();
    ldg<<<1, 1>>>(Result, Input);
    hipDeviceSynchronize();
    checkResult("__ldg", {TestCase.first}, TestCase.first, *Result,
                TestCase.second);
  }
}

__global__ void ldg(float *const Result, __hip_bfloat162 *Input1) {
  auto ret = __ldg(Input1);
  Result[0] = __bfloat162float(ret.x);
  Result[1] = __bfloat162float(ret.y);
}

void testLdgCases(const vector<pair<__hip_bfloat162, int>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    __hip_bfloat162 *Input;
    hipMallocManaged(&Input, sizeof(*Input));
    setValue<<<1, 1>>>(Input, TestCase.first);
    hipDeviceSynchronize();
    ldg<<<1, 1>>>(Result, Input);
    hipDeviceSynchronize();
    checkResult("__ldg", {TestCase.first}, TestCase.first,
                {Result[0], Result[1]}, TestCase.second);
  }
}

__global__ void ldlu(float *const Result, __hip_bfloat16 *Input1) {
  *Result = __ldlu(Input1);
}

void testLdluCases(const vector<pair<__hip_bfloat16, int>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    __hip_bfloat16 *Input;
    hipMallocManaged(&Input, sizeof(*Input));
    setValue<<<1, 1>>>(Input, TestCase.first);
    hipDeviceSynchronize();
    ldlu<<<1, 1>>>(Result, Input);
    hipDeviceSynchronize();
    checkResult("__ldlu", {TestCase.first}, TestCase.first, *Result,
                TestCase.second);
  }
}

__global__ void ldlu(float *const Result, __hip_bfloat162 *Input1) {
  auto ret = __ldlu(Input1);
  Result[0] = __bfloat162float(ret.x);
  Result[1] = __bfloat162float(ret.y);
}

void testLdluCases(const vector<pair<__hip_bfloat162, int>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    __hip_bfloat162 *Input;
    hipMallocManaged(&Input, sizeof(*Input));
    setValue<<<1, 1>>>(Input, TestCase.first);
    hipDeviceSynchronize();
    ldlu<<<1, 1>>>(Result, Input);
    hipDeviceSynchronize();
    checkResult("__ldlu", {TestCase.first}, TestCase.first,
                {Result[0], Result[1]}, TestCase.second);
  }
}

__global__ void stcg(float *const Result, __hip_bfloat16 Input1,
                     __hip_bfloat16 *const Temp) {
  __stcg(Temp, Input1);
  *Result = __bfloat162float(*Temp);
}

void testStcgCases(const vector<pair<__hip_bfloat16, int>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  __hip_bfloat16 *Temp;
  hipMallocManaged(&Temp, sizeof(*Temp));
  for (const auto &TestCase : TestCases) {
    stcg<<<1, 1>>>(Result, TestCase.first, Temp);
    hipDeviceSynchronize();
    checkResult("__stcg", {TestCase.first}, TestCase.first, *Result,
                TestCase.second);
  }
}

__global__ void stcg(float *const Result, __hip_bfloat162 Input1,
                     __hip_bfloat162 *const Temp) {
  __stcg(Temp, Input1);
  Result[0] = __bfloat162float(Temp->x);
  Result[1] = __bfloat162float(Temp->y);
}

void testStcgCases(const vector<pair<__hip_bfloat162, int>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  __hip_bfloat162 *Temp;
  hipMallocManaged(&Temp, sizeof(*Temp));
  for (const auto &TestCase : TestCases) {
    stcg<<<1, 1>>>(Result, TestCase.first, Temp);
    hipDeviceSynchronize();
    checkResult("__stcg", {TestCase.first}, TestCase.first,
                {Result[0], Result[1]}, TestCase.second);
  }
}

__global__ void stcs(float *const Result, __hip_bfloat16 Input1,
                     __hip_bfloat16 *const Temp) {
  __stcs(Temp, Input1);
  *Result = __bfloat162float(*Temp);
}

void testStcsCases(const vector<pair<__hip_bfloat16, int>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  __hip_bfloat16 *Temp;
  hipMallocManaged(&Temp, sizeof(*Temp));
  for (const auto &TestCase : TestCases) {
    stcs<<<1, 1>>>(Result, TestCase.first, Temp);
    hipDeviceSynchronize();
    checkResult("__stcs", {TestCase.first}, TestCase.first, *Result,
                TestCase.second);
  }
}

__global__ void stcs(float *const Result, __hip_bfloat162 Input1,
                     __hip_bfloat162 *const Temp) {
  __stcs(Temp, Input1);
  Result[0] = __bfloat162float(Temp->x);
  Result[1] = __bfloat162float(Temp->y);
}

void testStcsCases(const vector<pair<__hip_bfloat162, int>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  __hip_bfloat162 *Temp;
  hipMallocManaged(&Temp, sizeof(*Temp));
  for (const auto &TestCase : TestCases) {
    stcs<<<1, 1>>>(Result, TestCase.first, Temp);
    hipDeviceSynchronize();
    checkResult("__stcs", {TestCase.first}, TestCase.first,
                {Result[0], Result[1]}, TestCase.second);
  }
}

__global__ void stwb(float *const Result, __hip_bfloat16 Input1,
                     __hip_bfloat16 *const Temp) {
  __stwb(Temp, Input1);
  *Result = __bfloat162float(*Temp);
}

void testStwbCases(const vector<pair<__hip_bfloat16, int>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  __hip_bfloat16 *Temp;
  hipMallocManaged(&Temp, sizeof(*Temp));
  for (const auto &TestCase : TestCases) {
    stwb<<<1, 1>>>(Result, TestCase.first, Temp);
    hipDeviceSynchronize();
    checkResult("__stwb", {TestCase.first}, TestCase.first, *Result,
                TestCase.second);
  }
}

__global__ void stwb(float *const Result, __hip_bfloat162 Input1,
                     __hip_bfloat162 *const Temp) {
  __stwb(Temp, Input1);
  Result[0] = __bfloat162float(Temp->x);
  Result[1] = __bfloat162float(Temp->y);
}

void testStwbCases(const vector<pair<__hip_bfloat162, int>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  __hip_bfloat162 *Temp;
  hipMallocManaged(&Temp, sizeof(*Temp));
  for (const auto &TestCase : TestCases) {
    stwb<<<1, 1>>>(Result, TestCase.first, Temp);
    hipDeviceSynchronize();
    checkResult("__stwb", {TestCase.first}, TestCase.first,
                {Result[0], Result[1]}, TestCase.second);
  }
}

__global__ void stwt(float *const Result, __hip_bfloat16 Input1,
                     __hip_bfloat16 *const Temp) {
  __stwt(Temp, Input1);
  *Result = __bfloat162float(*Temp);
}

void testStwtCases(const vector<pair<__hip_bfloat16, int>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  __hip_bfloat16 *Temp;
  hipMallocManaged(&Temp, sizeof(*Temp));
  for (const auto &TestCase : TestCases) {
    stwt<<<1, 1>>>(Result, TestCase.first, Temp);
    hipDeviceSynchronize();
    checkResult("__stwt", {TestCase.first}, TestCase.first, *Result,
                TestCase.second);
  }
}

__global__ void stwt(float *const Result, __hip_bfloat162 Input1,
                     __hip_bfloat162 *const Temp) {
  __stwt(Temp, Input1);
  Result[0] = __bfloat162float(Temp->x);
  Result[1] = __bfloat162float(Temp->y);
}

void testStwtCases(const vector<pair<__hip_bfloat162, int>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  __hip_bfloat162 *Temp;
  hipMallocManaged(&Temp, sizeof(*Temp));
  for (const auto &TestCase : TestCases) {
    stwt<<<1, 1>>>(Result, TestCase.first, Temp);
    hipDeviceSynchronize();
    checkResult("__stwt", {TestCase.first}, TestCase.first,
                {Result[0], Result[1]}, TestCase.second);
  }
}

int main() {
  testBFloat1622float2Cases({
      {{-0.3, -0.5}, {{-0.30078125, -0.5}, 16}},
      {{0.3, 0.5}, {{0.30078125, 0.5}, 16}},
      {{30, 50}, {{30, 50}, 14}},
      {{0.432643, 0.23654}, {{0.43359375, 0.236328125}, 16}},
  });
  testBFloat162floatCases({
      {-0.3, {-0.30078125, 16}},
      {0.3, {0.30078125, 16}},
      {30, {30, 14}},
      {0.432643, {0.43359375, 16}},
  });
  testFloat22bFloat162_rnCases({
      {{-0.3, -0.5}, {{-0.30078125, -0.5}, 16}},
      {{0.3, 0.5}, {{0.30078125, 0.5}, 16}},
      {{30, 50}, {{30, 50}, 14}},
      {{0.432643, 0.23654}, {{0.43359375, 0.236328125}, 16}},
  });
  testFloat2bFloat16Cases({
      {-0.3, {-0.30078125, 16}},
      {0.3, {0.30078125, 16}},
      {30, {30, 14}},
      {0.432643, {0.43359375, 16}},
  });
  testLdcaCases({
      {-0.3, 16},
      {-0.4, 16},
      {0, 37},
      {0.7, 16},
      {1, 15},
      {100.6, 14},
  });
  testLdcaCases({
      {{-0.3, -0.4}, 16},
      {{0, 0.7}, 16},
      {{1, 100.6}, 14},
      {{100.6, 1}, 14},
  });
  testLdcgCases({
      {-0.3, 16},
      {-0.4, 16},
      {0, 37},
      {0.7, 16},
      {1, 15},
      {100.6, 14},
  });
  testLdcgCases({
      {{-0.3, -0.4}, 16},
      {{0, 0.7}, 16},
      {{1, 100.6}, 14},
      {{100.6, 1}, 14},
  });
  testLdcsCases({
      {-0.3, 16},
      {-0.4, 16},
      {0, 37},
      {0.7, 16},
      {1, 15},
      {100.6, 14},
  });
  testLdcsCases({
      {{-0.3, -0.4}, 16},
      {{0, 0.7}, 16},
      {{1, 100.6}, 14},
      {{100.6, 1}, 14},
  });
  testLdcvCases({
      {-0.3, 16},
      {-0.4, 16},
      {0, 37},
      {0.7, 16},
      {1, 15},
      {100.6, 14},
  });
  testLdcvCases({
      {{-0.3, -0.4}, 16},
      {{0, 0.7}, 16},
      {{1, 100.6}, 14},
      {{100.6, 1}, 14},
  });
  testLdgCases({
      {-0.3, 16},
      {-0.4, 16},
      {0, 37},
      {0.7, 16},
      {1, 15},
      {100.6, 14},
  });
  testLdgCases({
      {{-0.3, -0.4}, 16},
      {{0, 0.7}, 16},
      {{1, 100.6}, 14},
      {{100.6, 1}, 14},
  });
  testLdluCases({
      {-0.3, 16},
      {-0.4, 16},
      {0, 37},
      {0.7, 16},
      {1, 15},
      {100.6, 14},
  });
  testLdluCases({
      {{-0.3, -0.4}, 16},
      {{0, 0.7}, 16},
      {{1, 100.6}, 14},
      {{100.6, 1}, 14},
  });
  testStcgCases({
      {-0.3, 16},
      {-0.4, 16},
      {0, 37},
      {0.7, 16},
      {1, 15},
      {100.6, 14},
  });
  testStcgCases({
      {{-0.3, -0.4}, 16},
      {{0, 0.7}, 16},
      {{1, 100.6}, 14},
      {{100.6, 1}, 14},
  });
  testStcsCases({
      {-0.3, 16},
      {-0.4, 16},
      {0, 37},
      {0.7, 16},
      {1, 15},
      {100.6, 14},
  });
  testStcsCases({
      {{-0.3, -0.4}, 16},
      {{0, 0.7}, 16},
      {{1, 100.6}, 14},
      {{100.6, 1}, 14},
  });
  testStwbCases({
      {-0.3, 16},
      {-0.4, 16},
      {0, 37},
      {0.7, 16},
      {1, 15},
      {100.6, 14},
  });
  testStwbCases({
      {{-0.3, -0.4}, 16},
      {{0, 0.7}, 16},
      {{1, 100.6}, 14},
      {{100.6, 1}, 14},
  });
  testStwtCases({
      {-0.3, 16},
      {-0.4, 16},
      {0, 37},
      {0.7, 16},
      {1, 15},
      {100.6, 14},
  });
  testStwtCases({
      {{-0.3, -0.4}, 16},
      {{0, 0.7}, 16},
      {{1, 100.6}, 14},
      {{100.6, 1}, 14},
  });
  cout << "passed " << passed << "/" << passed + failed << " cases!" << endl;
  if (failed) {
    cout << "failed!" << endl;
  }
  return failed;
}
